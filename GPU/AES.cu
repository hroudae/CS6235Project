#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include "constants.h"
#include "AES_lib.h"
#include "fileio.h"


#define KEY_SIZE_ARGUMENT_INDEX     1
#define KEY_FP_INDEX                2
#define PLAIN_TEXT_FP_INDEX         3
#define MODE_INDEX                  4

#define CHARS_PER_BYTE              2


__global__ void
naive_AES_encrypt(uint8_t* cipherText_d, uint8_t* plainText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i<numPlainTextBlocks)
    {
        AES_Encrypt_Block(plainText_d  + i * (BLOCK_SIZE_BITS / 8),
                          cipherText_d + i * (BLOCK_SIZE_BITS / 8),
                          roundKeys_d, numRounds);
    }
}

__global__ void
naive_AES_decrypt(uint8_t* cipherText_d, uint8_t* plainText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i<numPlainTextBlocks)
    {
        AES_Decrypt_Block(cipherText_d + i * (BLOCK_SIZE_BITS / 8),
                          plainText_d  + i * (BLOCK_SIZE_BITS / 8),
                          roundKeys_d, numRounds);
    }

}

__global__ void
ctr_AES_encrypt(uint8_t* cipherText_d, uint8_t* plainText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks, uint8_t* counter) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int j;

    if(i<numPlainTextBlocks)
    {
        uint8_t ctr[16];
        incrementCounter(ctr, counter, i);
        AES_Encrypt_Block(ctr,
                          cipherText_d + i * (BLOCK_SIZE_BITS / 8),
                          roundKeys_d, numRounds);

        for (j = 0; j < 16; j++) {
            *((cipherText_d+i*(BLOCK_SIZE_BITS / 8))+j) ^= *(plainText_d+i*(BLOCK_SIZE_BITS / 8)+j);
        }
    }
}

__global__ void
ctr_AES_decrypt(uint8_t* cipherText_d, uint8_t* plainText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks, uint8_t* counter) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int j;

    if(i<numPlainTextBlocks)
    {
        uint8_t ctr[16];
        incrementCounter(ctr, counter, i);
        AES_Encrypt_Block(ctr,
            plainText_d + i * (BLOCK_SIZE_BITS / 8),
            roundKeys_d, numRounds);

        for (j = 0; j < 16; j++)
            *(plainText_d+i*(BLOCK_SIZE_BITS / 8)+j) ^= *((cipherText_d+i*(BLOCK_SIZE_BITS / 8))+j);
    }
}

//NOT TESTED
__global__ void
cbc_AES_encrypt(uint8_t* cipherText_d, uint8_t* plainText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks, uint8_t* counter) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int j;

    int step = blockDim.x * blockDim.y * blockDim.z; // Should be 256 right now. Might need to mult by num blocks
    int initial = 0; //Janky way to do first step. Theres prob a cleaner mod math way but /shrug

    uint8_t ctr[16]; //Treating ctr as init vector. Moved up to avoid re-init
    incrementCounter(ctr, counter, i); //toss some values in there

    //Doesn't this process only happen once in other versions?
    //Need as many threads as you have blocks of data
    //Designing mine to assume that data blocks >>>> thread limit (1024 max)
    //But current setup is 256 threads so gonna roll w/ that
    while(i<numPlainTextBlocks)
    {
        if (initial != 0){ //Conditionals are bad in kernels aren't they. Wrapped to avoid doing 16 chex
            for (j = 0; j < 16; j++) {
                *(plainText_d+i*(BLOCK_SIZE_BITS / 8)+j) ^= *((cipherText_d+(i-1)*(BLOCK_SIZE_BITS / 8))+j);
            }
        }
      AES_Encrypt_Block(plainText_d+i*(BLOCK_SIZE_BITS / 8),
                        cipherText_d+i*(BLOCK_SIZE_BITS / 8),
                        roundKeys_d, numRounds);
        i+= step;
    }
}

static hipError_t AES_Encrypt(uint8_t* plainText_h, uint8_t* cipherText_h, uint32_t* roundKeys_h, NumRounds_t numRounds, uint32_t plainTextSize_bytes, ModeOfOperation_t mode, uint8_t *iv_h)
{
    hipError_t err       = hipSuccess;
    uint8_t* plainText_d  = NULL;
    uint8_t* cipherText_d = NULL;
    uint32_t* roundKeys_d = NULL;
    uint32_t plainTextBlockCnt;
    uint8_t* iv_d = NULL;

    hipEvent_t start, stop;
    float seconds = 0;


    hipEventCreate(&start);
    hipEventCreate(&stop);


    /*** Malloc Device memory ***/
    err = hipMalloc((void**)&plainText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector plainText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&roundKeys_d, sizeof(uint32_t) * numRounds * 4);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector roundKeys_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&cipherText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cipherText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (mode == CTR) {
        err = hipMalloc((void**)&iv_d, sizeof(uint8_t)*16);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device IV_d (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    /*** Copy Data from Host to Device memory ***/
    err = hipMemcpy(plainText_d, plainText_h, plainTextSize_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector plainText from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(roundKeys_d, roundKeys_h, sizeof(uint32_t) * numRounds * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector roundKeys from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // generate a random IV to be used in CTR mode
    if (mode == CTR) {
        if (GetIV(iv_h) < 0) {
            printf("Error getting IV!\n");
            exit(EXIT_FAILURE);
        }
        err = hipMemcpy(iv_d, iv_h, sizeof(uint8_t) * 16, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy IV from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    plainTextBlockCnt = (plainTextSize_bytes + (BLOCK_SIZE_BITS / 8)-1) / (BLOCK_SIZE_BITS / 8);

    int threadBlockDim = 256;
    dim3 threadsPerBlock(threadBlockDim, 1, 1);
    dim3 blocksPerGrid((plainTextSize_bytes+threadBlockDim-1)/threadBlockDim, 1, 1);

    hipEventRecord(start);
    if (mode == CTR)
        ctr_AES_encrypt<<<blocksPerGrid, threadsPerBlock>>>(cipherText_d, plainText_d, roundKeys_d, numRounds, plainTextBlockCnt, iv_d);
    else
        naive_AES_encrypt<<<blocksPerGrid, threadsPerBlock>>>(cipherText_d, plainText_d, roundKeys_d, numRounds, plainTextBlockCnt);
    hipEventRecord(stop);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch AES kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(cipherText_h, cipherText_d, plainTextSize_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cipherText from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventSynchronize(stop);
    hipEventElapsedTime(&seconds, start, stop);

    fprintf(stderr, "Encrypt Execution Time: %fs\n", seconds);


    /*** Free Device Mem ***/
    err = hipFree(plainText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector plainText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(roundKeys_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector roundKeys (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cipherText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector cipherText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (mode == CTR) {
        err = hipFree(iv_d);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector IV (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // TODO: Do we reset the device here or only at the end of main?

    return err;
}

hipError_t AES_Decrypt(uint8_t* plainText_h, uint8_t* cipherText_h, uint32_t* roundKeys_h, NumRounds_t numRounds, uint32_t plainTextSize_bytes, ModeOfOperation_t mode, uint8_t *iv_h)
{
    hipError_t err       = hipSuccess;
    uint32_t* roundKeys_d = NULL;
    uint8_t* plainText_d  = NULL;
    uint8_t* cipherText_d = NULL;
    uint32_t plainTextBlockCnt;
    uint8_t* iv_d = NULL;

    hipEvent_t start, stop;
    float seconds = 0;


    hipEventCreate(&start);
    hipEventCreate(&stop);


    /*** Malloc Device memory ***/
    err = hipMalloc((void**)&plainText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector plainText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&roundKeys_d, sizeof(uint32_t) * numRounds * 4);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector roundKeys_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&cipherText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cipherText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (mode == CTR) {
        err = hipMalloc((void**)&iv_d, sizeof(uint8_t)*16);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device IV_d (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    /*** Copy Data from Host to Device memory ***/
    err = hipMemcpy(cipherText_d, cipherText_h, plainTextSize_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cipherText from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(roundKeys_d, roundKeys_h, sizeof(uint32_t) * numRounds * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector roundKeys from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (mode == CTR) {
        err = hipMemcpy(iv_d, iv_h, sizeof(uint8_t) * 16, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy IV from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    plainTextBlockCnt = (plainTextSize_bytes + (BLOCK_SIZE_BITS / 8)-1) / (BLOCK_SIZE_BITS / 8);

    int threadBlockDim = 256;
    dim3 threadsPerBlock(threadBlockDim, 1, 1);
    dim3 blocksPerGrid((plainTextSize_bytes+threadBlockDim-1)/threadBlockDim, 1, 1);

    hipEventRecord(start);
    if (mode == CTR)
        ctr_AES_decrypt<<<blocksPerGrid, threadsPerBlock>>>(cipherText_d, plainText_d, roundKeys_d, numRounds, plainTextBlockCnt, iv_d);
    else
        naive_AES_decrypt<<<blocksPerGrid, threadsPerBlock>>>(cipherText_d, plainText_d, roundKeys_d, numRounds, plainTextBlockCnt);
    hipEventRecord(stop);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch AES kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(plainText_h, plainText_d, plainTextSize_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector plainText from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventSynchronize(stop);
    hipEventElapsedTime(&seconds, start, stop);

    fprintf(stderr, "Decrypt Execution Time: %fs\n", seconds);


    /*** Free Device Mem ***/
    err = hipFree(plainText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector plainText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(roundKeys_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector roundKeys (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cipherText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector cipherText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (mode == CTR) {
        err = hipFree(iv_d);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector IV (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // TODO: Do we reset the device here or only at the end of main?

    return err;
}

/* arguments keySize, keyFile, plainTextFile, mode*/
/* mode is 0 for ECB, 1 for CTR */
main( int argc, char **argv )
{
    hipError_t err = hipSuccess;
    uint32_t plainTextSize_bytes = 0;
    uint32_t plainTextSizeAligned_bytes = 0;
    uint32_t loopNdx;

    uint32_t* key;
    uint32_t* roundKeys;

    KeySize_Word_t keySize_words = AES128_KEYSIZE;
    NumRounds_t rounds = AES128_ROUNDS;
    AESVersion_t version = AES128_VERSION;
    ModeOfOperation_t mode = ECB;

    uint8_t* en_plainText;
    uint8_t* de_plainText;
    uint8_t* plainText_verification;
    uint8_t* cipherText;
    uint8_t *iv = (uint8_t*)calloc(sizeof(uint8_t) * 16, sizeof(uint8_t));

    bool verificationSuccessful = true;

#ifdef USE_TEST_CODE
    uint32_t appendedZeroCnt_bytes = 0;
#else
    unsigned char* inFilekey;
    uint32_t expectedKeySize;
    unsigned char* inputPlainText;
    uint32_t numCharRead = 0;
    uint32_t appendedZeroCnt_bytes = 0;
#endif


    if(argc > 1)
    {

#ifdef USE_TEST_CODE
        fprintf(stderr, "Test code enabled: Cannot supply arguments\n");
        return 1;
#else


        if(argc > 5)
        {
            fprintf(stderr, "Expecting at most 4 arguments: Keysize, KeyfilePath, PlainTextPath, Mode\n");
        }


        if(atoi(argv[KEY_SIZE_ARGUMENT_INDEX]) == BIT_KEY_128)
        {
            keySize_words = AES128_KEYSIZE;
            rounds        = AES128_ROUNDS;
            version       = AES128_VERSION;
        }
        else if(atoi(argv[KEY_SIZE_ARGUMENT_INDEX]) == BIT_KEY_192)
        {
            keySize_words = AES192_KEYSIZE;
            rounds        = AES192_ROUNDS;
            version       = AES192_VERSION;
        }
        else if(atoi(argv[KEY_SIZE_ARGUMENT_INDEX]) == BIT_KEY_256)
        {
            keySize_words = AES256_KEYSIZE;
            rounds        = AES256_ROUNDS;
            version       = AES256_VERSION;
        }
        else
        {
            fprintf(stderr, "Invalid key size: %d\n", atoi(argv[KEY_SIZE_ARGUMENT_INDEX]));
        }

        if (argc == 5 && atoi(argv[MODE_INDEX]) == ECB) {
            printf("ECB mode chosen.\n");
            mode = ECB;
        }
        else if (argc == 5 && atoi(argv[MODE_INDEX]) == CTR) {
            printf("CTR mode chosen.\n");
            mode = CTR;
        }
        else if (argc == 5) fprintf(stderr, "Invalid mode: %d\n", atoi(argv[MODE_INDEX]));
        else {
            printf("No mode provided, defaulting to ECB\n");
            mode = ECB;
        }


        expectedKeySize = keySize_words*sizeof(uint32_t)*CHARS_PER_BYTE;

        numCharRead = readfile(argv[KEY_FP_INDEX], &inFilekey, expectedKeySize);
        if (numCharRead < 1 || numCharRead != expectedKeySize)
        {
            fprintf(stderr, "ERROR reading key file with size: %d\n", numCharRead);
            return 1;
        }
        else
        {
            fprintf(stderr, "Read %d bytes from input key file\n", numCharRead/CHARS_PER_BYTE);
        }


        plainTextSize_bytes = readfile(argv[PLAIN_TEXT_FP_INDEX], &inputPlainText, 16777216);
        if (plainTextSize_bytes < 1)
        {
            fprintf(stderr, "ERROR reading plainText file\n");
            return 1;
        }
        else
        {
            fprintf(stderr, "Read %d bytes from input plain text file\n", plainTextSize_bytes);
        }

#if 0
        fprintf(stderr, "\n");
        for(loopNdx=0; loopNdx<100; loopNdx++)
        {
            fprintf(stderr,"%c", inputPlainText[loopNdx]);
        }
        fprintf(stderr, "\n");
#endif

        fprintf(stderr, "\n");
#endif
    }
    else
    {
#ifdef USE_TEST_CODE
        fprintf(stderr, "Using hardcoded test: 1 block and 256 bit key\n");
#else
        fprintf(stderr, "insufficient arguments and test code disabled.\n");
#endif
    }


#ifdef USE_TEST_CODE
    keySize_words       = AES256_KEYSIZE;
    rounds              = AES256_ROUNDS;
    plainTextSize_bytes = 16;
#endif

    appendedZeroCnt_bytes = BLOCK_SIZE_BITS - plainTextSize_bytes%BLOCK_SIZE_BITS;
    plainTextSizeAligned_bytes = plainTextSize_bytes + appendedZeroCnt_bytes;

    key = (uint32_t*)calloc(sizeof(uint32_t*) * keySize_words, sizeof(uint32_t));
    roundKeys = (uint32_t*)calloc(sizeof(uint32_t*) * rounds * 4, sizeof(uint32_t));
    en_plainText = (unsigned char*)calloc(sizeof(unsigned char) * plainTextSizeAligned_bytes, sizeof(uint8_t));
    de_plainText = (unsigned char*)calloc(sizeof(unsigned char) * plainTextSizeAligned_bytes, sizeof(uint8_t));
    plainText_verification = (unsigned char*)calloc(sizeof(unsigned char) * plainTextSize_bytes, sizeof(uint8_t));
    cipherText = (unsigned char*)calloc(sizeof(unsigned char) * plainTextSizeAligned_bytes, sizeof(uint8_t));

#ifdef USE_TEST_CODE
    uint32_t sample256Key[8] = {0x00010203, 0x04050607, 0x08090a0b, 0x0c0d0e0f,
                                0x10111213, 0x14151617, 0x18191a1b, 0x1c1d1e1f};

    uint8_t sampleDataBlock[16] = {0x00, 0x11, 0x22, 0x33, 0x44, 0x55, 0x66, 0x77,
                                   0x88, 0x99, 0xaa, 0xbb, 0xcc, 0xdd, 0xee, 0xff};

    memcpy((void*)key, (void*)sample256Key, sizeof(uint32_t*)*keySize_words);
    memcpy((void*)en_plainText, (void*)sampleDataBlock, plainTextSize_bytes);
    memcpy((void*)plainText_verification, (void*)sampleDataBlock, plainTextSize_bytes);

#else
    // TODO: copy supplied key file into key
    /*uint32_t inputKey[8] = {0x00010203, 0x04050607, 0x08090a0b, 0x0c0d0e0f,
                                0x10111213, 0x14151617, 0x18191a1b, 0x1c1d1e1f};*/

    getDecKeyfromAsciiKey((char*)inFilekey, key, keySize_words);

#if 0
    fprintf(stderr, "\n");
    for(loopNdx=0; loopNdx<keySize_words; loopNdx++)
    {
        fprintf(stderr,"key[%d]=0x%08x\n", loopNdx, key[loopNdx]);
    }
    fprintf(stderr, "\n");
#endif

    memcpy((void*)en_plainText, (void*)inputPlainText, plainTextSize_bytes);
    memcpy((void*)plainText_verification, (void*)inputPlainText, plainTextSize_bytes);
#endif

    KeyExpansion(key, roundKeys, version);

#if 0
    for(loopNdx=0; loopNdx<plainTextSizeAligned_bytes; loopNdx++)
    {
        printf("plaintText[%d]=%02x\n", loopNdx, en_plainText[loopNdx]);
    }
    fprintf(stderr, "\n");
#endif
    err = AES_Encrypt(en_plainText, cipherText, roundKeys, rounds, plainTextSizeAligned_bytes, mode, iv);

#if 0
    for(loopNdx=0; loopNdx<plainTextSizeAligned_bytes; loopNdx++)
    {
        printf("cipherText[%d]=%02x\n", loopNdx, cipherText[loopNdx]);
    }
    fprintf(stderr, "\n");
#endif

    err = AES_Decrypt(de_plainText, cipherText, roundKeys, rounds, plainTextSizeAligned_bytes, mode, iv);

#if 0
    fprintf(stderr, "Verifications plainTextSize_bytes: %d\n", plainTextSize_bytes);
    fprintf(stderr, "Verifications plainTextSizeAligned_bytes: %d\n", plainTextSizeAligned_bytes);
#endif

    for(loopNdx=0; loopNdx<plainTextSize_bytes; loopNdx++)
    {
        if(de_plainText[loopNdx] != plainText_verification[loopNdx])
        {
            fprintf(stderr, "Verification Failed at index %d! %02x!=%02x\n",
                loopNdx, de_plainText[loopNdx], plainText_verification[loopNdx]);

            verificationSuccessful = false;
        }
    }

    if(verificationSuccessful)
    {
        fprintf(stderr, "\nVerification successful\n");
    }

    /*** Free Host Memory ***/
    free(key);
    free(roundKeys);
    free(iv);


    // TODO: What do we do with the data? (write to a file, compare against expected, return, etc)

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    fprintf(stderr, "AES Execution Completed\n");
}
