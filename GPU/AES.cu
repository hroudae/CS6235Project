#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include "constants.h"
#include "AES_lib.h"

#define USE_TEST_CODE               1

#define KEY_SIZE_ARGUMENT_INDEX     1




__global__ void
naive_AES_encrypt(uint8_t* cipherText_d, uint8_t* plainText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i<numPlainTextBlocks)
    {
        AES_Encrypt_Block(plainText_d  + i * (BLOCK_SIZE_BITS / 8), 
                          cipherText_d + i * (BLOCK_SIZE_BITS / 8), 
                          roundKeys_d, numRounds);
    }
}

__global__ void
naive_AES_decrypt(uint8_t* plainText_d, uint8_t* cipherText_d, uint32_t* roundKeys_d, NumRounds_t numRounds, uint32_t numPlainTextBlocks)
{
    //int i = blockDim.x*blockIdx.x + threadIdx.x;

   /* if(i<numPlainTextBlocks)
    {
        AES_Decrypt_Block(cipherText_d + i * (BLOCK_SIZE_BITS / 8), 
                          plainText_d  + i * (BLOCK_SIZE_BITS / 8), 
                          roundKeys_d, numRounds);
    }   */

}

static hipError_t AES_Encrypt(uint8_t* plainText_h, uint8_t* cipherText_h, uint32_t* roundKeys_h, NumRounds_t numRounds, uint32_t plainTextSize_bytes)
{
    hipError_t err       = hipSuccess;
    uint8_t* plainText_d  = NULL;
    uint8_t* cipherText_d = NULL;
    uint32_t* roundKeys_d = NULL;
    uint32_t plainTextBlockCnt;


    /*** Malloc Device memory ***/
    err = hipMalloc((void**)&plainText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector plainText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&roundKeys_d, sizeof(uint32_t) * numRounds * 4);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector roundKeys_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&cipherText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cipherText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*** Copy Data from Host to Device memory ***/
    err = hipMemcpy(plainText_d, plainText_h, plainTextSize_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector plainText from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(roundKeys_d, roundKeys_h, sizeof(uint32_t) * numRounds * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector roundKeys from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    plainTextBlockCnt = (plainTextSize_bytes + (BLOCK_SIZE_BITS / 8)-1) / (BLOCK_SIZE_BITS / 8);

    int threadBlockDim = 256;
    dim3 threadsPerBlock(threadBlockDim, 1, 1);
    dim3 blocksPerGrid((plainTextSize_bytes+threadBlockDim-1)/threadBlockDim, 1, 1);

    naive_AES_encrypt<<<blocksPerGrid, threadsPerBlock>>>(cipherText_d, plainText_d, roundKeys_d, numRounds, plainTextBlockCnt);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch AES kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(cipherText_h, cipherText_d, plainTextSize_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cipherText from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    /*** Free Device Mem ***/
    err = hipFree(plainText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector plainText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(roundKeys_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector roundKeys (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cipherText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector cipherText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // TODO: Do we reset the device here or only at the end of main?

    return err;
}

hipError_t AES_Decrypt(uint8_t* plainText_h, uint8_t* cipherText_h, uint32_t* roundKeys_h, NumRounds_t numRounds, uint32_t plainTextSize_bytes)
{
    hipError_t err       = hipSuccess;
    uint32_t* roundKeys_d = NULL;
    uint8_t* plainText_d  = NULL;
    uint8_t* cipherText_d = NULL;
    uint32_t plainTextBlockCnt;


    /*** Malloc Device memory ***/
    err = hipMalloc((void**)&plainText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector plainText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&roundKeys_d, sizeof(uint32_t) * numRounds * 4);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector roundKeys_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&cipherText_d, plainTextSize_bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cipherText_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*** Copy Data from Host to Device memory ***/
    err = hipMemcpy(cipherText_d, cipherText_h, plainTextSize_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cipherText from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(roundKeys_d, roundKeys_h, sizeof(uint32_t) * numRounds * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector roundKeys from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    plainTextBlockCnt = (plainTextSize_bytes + (BLOCK_SIZE_BITS / 8)-1) / (BLOCK_SIZE_BITS / 8);

    int threadBlockDim = 256;
    dim3 threadsPerBlock(threadBlockDim, 1, 1);
    dim3 blocksPerGrid((plainTextSize_bytes+threadBlockDim-1)/threadBlockDim, 1, 1);

    naive_AES_encrypt<<<blocksPerGrid, threadsPerBlock>>>(cipherText_d, plainText_d, roundKeys_d, numRounds, plainTextBlockCnt);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch AES kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(plainText_h, plainText_d, plainTextSize_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector plainText from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    /*** Free Device Mem ***/
    err = hipFree(plainText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector plainText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(roundKeys_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector roundKeys (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cipherText_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector cipherText (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // TODO: Do we reset the device here or only at the end of main?

    return err;
}


main( int argc, char **argv )
{
    hipError_t err = hipSuccess;
    uint32_t plainTextSize_bytes = 0;

    uint32_t* key;
    uint32_t* roundKeys;

    KeySize_Word_t keySize_words = AES128_KEYSIZE;
    NumRounds_t rounds = AES128_ROUNDS;
    AESVersion_t version = AES128_VERSION;

    uint8_t* en_plainText;
    uint8_t* de_plainText;
    uint8_t* plainText_verification;
    uint8_t* cipherText;

    // TODO: Need to also supply filename of key 
    if(argc > 1)
    {
        if(atoi(argv[KEY_SIZE_ARGUMENT_INDEX]) == BIT_KEY_128)
        {
            keySize_words = AES128_KEYSIZE;
            rounds        = AES128_ROUNDS;
            version       = AES128_VERSION;
        }
        else if(atoi(argv[KEY_SIZE_ARGUMENT_INDEX]) == BIT_KEY_192)
        {
            keySize_words = AES192_KEYSIZE;
            rounds        = AES192_ROUNDS;
            version       = AES192_VERSION;
        }
        else if(atoi(argv[KEY_SIZE_ARGUMENT_INDEX]) == BIT_KEY_256)
        {
            keySize_words = AES256_KEYSIZE;
            rounds        = AES256_ROUNDS;
            version       = AES256_VERSION;
        }
        else
        {
            fprintf(stderr, "Invalid key size: %d\n", atoi(argv[KEY_SIZE_ARGUMENT_INDEX]));
        }
    }


#if (USE_TEST_CODE)
    keySize_words       = AES256_KEYSIZE;
    rounds              = AES256_ROUNDS;
    plainTextSize_bytes = 16;
#endif


    key = (uint32_t*)malloc(sizeof(uint32_t*) * keySize_words);
    roundKeys = (uint32_t*)malloc(sizeof(uint32_t*) * rounds * 4);
    en_plainText = (unsigned char*)malloc(sizeof(unsigned char) * plainTextSize_bytes);
    de_plainText = (unsigned char*)malloc(sizeof(unsigned char) * plainTextSize_bytes);
    plainText_verification = (unsigned char*)malloc(sizeof(unsigned char) * plainTextSize_bytes);
    cipherText = (unsigned char*)malloc(sizeof(unsigned char) * plainTextSize_bytes);

#if (USE_TEST_CODE)
    uint32_t sample256Key[8] = {0x00010203, 0x04050607, 0x08090a0b, 0x0c0d0e0f, 
                                0x10111213, 0x14151617, 0x18191a1b, 0x1c1d1e1f};

    uint8_t sampleDataBlock[16] = {0x00, 0x11, 0x22, 0x33, 0x44, 0x55, 0x66, 0x77, 
                                   0x88, 0x99, 0xaa, 0xbb, 0xcc, 0xdd, 0xee, 0xff};

    memcpy((void*)key, (void*)sample256Key, sizeof(uint32_t*)*keySize_words);
    memcpy((void*)en_plainText, (void*)sampleDataBlock, plainTextSize_bytes);
    memcpy((void*)plainText_verification, (void*)sampleDataBlock, plainTextSize_bytes);

#else
    // TODO: copy supplied key file into key
    memcpy((void*)key, (void*)inputKey, sizeof(uint32_t*)*keySize_words);
    memcpy((void*)en_plainText, (void*)inputPlainText, plainTextSize_bytes);
    memcpy((void*)plainText_verification, (void*)inputPlainText, plainTextSize_bytes);
#endif

    KeyExpansion(key, roundKeys, version);

    err = AES_Encrypt(en_plainText, cipherText, roundKeys, rounds, plainTextSize_bytes);

    err = AES_Decrypt(de_plainText, cipherText, roundKeys, rounds, plainTextSize_bytes);

    /*** Free Host Memory ***/
    free(key);
    free(roundKeys);


    // TODO: What do we do with the data? (write to a file, compare against expected, return, etc) 

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    fprintf(stderr, "GPU Implemntaion of AES Completed \n"); 
}
