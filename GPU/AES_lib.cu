#include "hip/hip_runtime.h"
#include "AES_lib.h"
#include "constants.h"

#include <stdint.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


/* Key Functions */
uint32_t rotl32 (uint32_t value, unsigned int count);
uint32_t SubWord(uint32_t word);

/* Encrypt Functions */
__device__ void AddRoundKey(state_t* state, uint32_t* roundKeys, int round);
__device__ void SubBytes(state_t* state);
__device__ void ShiftRows(state_t* state);
__device__ void MixColumns(state_t* state);

/* Decrypt Functions */
__device__ void InvSubBytes(state_t* state);
__device__ void InvShiftRows(state_t* state);
__device__ void InvMixColumns(state_t* state);


// Get random bytes from /dev/urandom 
int GetIV(uint8_t *iv) {
    FILE *f = fopen("/dev/urandom", "rb");
    if (f == NULL) { // couldn't open the file
        perror("ERROR");
        return -1;
    }
    else { // get 16 random bytes
        int rd = fread(iv, 1, 16, f);
        fclose(f);
        if (rd < 16) {
            perror("ERROR");
            return -1;
        }
    }
    return 0;
}

// Add the number to the 128 bit counter. the incremented number is in new_ctr
__device__
void incrementCounter(uint8_t *new_ctr, uint8_t *ctr, int inc) {
    uint64_t high = ((uint64_t) ctr[0] << 56) | ((uint64_t) ctr[1] << 48) | 
                    ((uint64_t) ctr[2] << 40) | ((uint64_t) ctr[3] << 32) |
                    ((uint64_t) ctr[4] << 24) | ((uint64_t) ctr[5] << 16) | 
                    ((uint64_t) ctr[6] << 8)  | ((uint64_t) ctr[7]);
    uint64_t low  = ((uint64_t) ctr[8] << 56)  | ((uint64_t) ctr[9] << 48)  | 
                    ((uint64_t) ctr[10] << 40) | ((uint64_t) ctr[11] << 32) |
                    ((uint64_t) ctr[12] << 24) | ((uint64_t) ctr[13] << 16) | 
                    ((uint64_t) ctr[14] << 8)  | ((uint64_t) ctr[15]);
    uint64_t count = (uint64_t) inc;
    uint64_t newlow = low + count;
    if (newlow < low) high++;
    int i, shift;
    for (i = 0, shift = 56; i < 8; i++, shift -= 8) {
        new_ctr[i] = (uint8_t) (high >> shift);
        new_ctr[i+8] = (uint8_t) (newlow >> shift);
    }
}

// circular shift left: https://en.wikipedia.org/wiki/Circular_shift
uint32_t rotl32 (uint32_t value, unsigned int count) {
    const unsigned int mask = CHAR_BIT * sizeof(value) - 1;
    count &= mask;
    return (value << count) | (value >> (-count & mask));
}


uint32_t SubWord(uint32_t word) {
    // substitute word using sbox
    uint8_t byte0 = word >> 24;
    uint8_t byte1 = word >> 16 & ~(0xFF << 8);
    uint8_t byte2 = word >> 8 & ~(0xFFFF << 8);
    uint8_t byte3 = word & ~(0xFFFFFF << 8);
    uint32_t sub = (sbox[byte0] << 24) | (sbox[byte1] << 16) | (sbox[byte2] << 8) | sbox[byte3];
    return sub;
}

// key and roundKeys are stored in array of 32-bit unsigned int
// roundKey has 4(Number rounds)-1 32-bit words

void KeyExpansion(uint32_t* key, uint32_t* roundKeys, AESVersion_t vers) {
    unsigned int numround = 0;
    unsigned int keysize = 0;

    switch(vers) {
        case AES128_VERSION:
            numround = AES128_ROUNDS;
            keysize = AES128_KEYSIZE;
            break;
        case AES192_VERSION:
            numround = AES192_ROUNDS;
            keysize = AES192_KEYSIZE;
            break;
        case AES256_VERSION:
            numround = AES256_ROUNDS;
            keysize = AES256_KEYSIZE;
            break;
        default:
            numround = 0;
            keysize = 0;
    }

    int i;
    for (i = 0; i < 4*numround; i++) {
        if (i < keysize){
            roundKeys[i] = key[i];
        }
        else if (i >= keysize && i % keysize == 0) {
            // rotate one-byte left-circular
            uint32_t rot = rotl32(roundKeys[i-1],8);
            uint32_t sub = SubWord(rot);
            uint32_t rcon = rc[(i/keysize)-1] << 24;
            roundKeys[i] = roundKeys[i-keysize] ^ sub ^ rcon;
        }
        else if(i >= keysize && keysize > 6 && i % keysize == 4) {
            uint32_t sub = SubWord(roundKeys[i-1]);
            roundKeys[i] = roundKeys[i-keysize] ^ sub;
        }
        else {
            roundKeys[i] = roundKeys[i-keysize] ^ roundKeys[i-1];
        }
    }
}

__device__
void AddRoundKey(state_t* state, uint32_t* roundKeys, int round) {
    int i, j, s;
    for (i = 0, s = 3; i < 4; i++, s--) {
        for (j = 0; j < 4; j ++) {
            // printf("%02x XOR %02x = ",(*state)[i][j], (uint8_t)(roundKeys[(round-1)*4 + j] >> (8*s)));
            (*state)[i][j] ^= (uint8_t) (roundKeys[(round-1)*4 + j] >> (8*s));
            // printf("%02x\n", (*state)[i][j]);
        }
    }
}

__device__
void SubBytes(state_t* state) {
    int i, j;
    for (i = 0; i < 4; i++,j--) {
        for (j = 0; j < 4; j ++) {
            (*state)[i][j] = sbox_d[(*state)[i][j]];
        }
    }
}

__device__
void ShiftRows(state_t* state) {
    uint8_t temp = (*state)[1][0];
    (*state)[1][0] = (*state)[1][1];
    (*state)[1][1] = (*state)[1][2];
    (*state)[1][2] = (*state)[1][3];
    (*state)[1][3] = temp;

    temp = (*state)[2][0];
    (*state)[2][0] = (*state)[2][2];
    (*state)[2][2] = temp;
    temp = (*state)[2][1];
    (*state)[2][1] = (*state)[2][3];
    (*state)[2][3] = temp;

    temp = (*state)[3][3];
    (*state)[3][3] = (*state)[3][2];
    (*state)[3][2] = (*state)[3][1];
    (*state)[3][1] = (*state)[3][0];
    (*state)[3][0] = temp;
}

__device__
void MixColumns(state_t* state) {
    int i, j;
    uint8_t col[4];
    uint8_t mult[4]; // each element of the column of state multiplied by 2
    for (j = 0; j < 4; j++) {
        for (i = 0; i < 4; i++) {
            col[i] = (*state)[i][j];
            uint8_t high = (col[i] >> 7) & 1;
            mult[i] = col[i] << 1; // multiply by 2
            mult[i] ^= high * 0x1b; // XOR with 0x1b if MSB was 1
        }

        // (*state)[0][j] = mult[0] ^ col[3] ^ col[2] ^ mult[1] ^ col[1];
        // (*state)[1][j] = mult[1] ^ col[0] ^ col[3] ^ mult[2] ^ col[2];
        // (*state)[2][j] = mult[2] ^ col[1] ^ col[0] ^ mult[3] ^ col[3];
        // (*state)[3][j] = mult[3] ^ col[2] ^ col[1] ^ mult[0] ^ col[0];

        (*state)[0][j] = mult_x2[col[0]] ^ mult_x3[col[1]] ^ col[2] ^ col[3];
        (*state)[1][j] = col[0] ^ mult_x2[col[1]] ^ mult_x3[col[2]] ^ col[3];
        (*state)[2][j] = col[0] ^ col[1] ^ mult_x2[col[2]] ^ mult_x3[col[3]];
        (*state)[3][j] = mult_x3[col[0]] ^ col[1] ^ col[2] ^ mult_x2[col[3]];
    }
}

// 128 bit chunk of data - 16 chars
__device__
void AES_Encrypt_Block(uint8_t* plainText, uint8_t* cipherText, uint32_t* roundKeys, unsigned int numround) 
{
    state_t state;
    int i, j, d;
    for (i = 0, d = 0; i < 4; i++)
        for (j = 0; j < 4; j++, d++) {
            state[j][i] = (uint8_t) plainText[d];
        }
    // round 1 - just add key
    AddRoundKey(&state, roundKeys, 1);

    // the rest of the rounds except the final
    int round;
    for (round = 2; round < numround; round++) {
        SubBytes(&state);
        ShiftRows(&state);
        MixColumns(&state);
        AddRoundKey(&state, roundKeys, round);
    }

    // final round
    SubBytes(&state);
    ShiftRows(&state);
    AddRoundKey(&state, roundKeys, numround);

    for (i = 0, d = 0; i < 4; i++)
        for (j = 0; j < 4; j++, d++) {
            cipherText[d] = (uint8_t) state[j][i];
        }
}

__device__
void InvSubBytes(state_t* state) {
    int i, j;
    for (i = 0; i < 4; i++,j--) {
        for (j = 0; j < 4; j ++) {
            (*state)[i][j] = invsbox[(*state)[i][j]];
        }
    }
}

__device__
void InvShiftRows(state_t* state) {
    uint8_t temp = (*state)[1][3];
    (*state)[1][3] = (*state)[1][2];
    (*state)[1][2] = (*state)[1][1];
    (*state)[1][1] = (*state)[1][0];
    (*state)[1][0] = temp;

    temp = (*state)[2][0];
    (*state)[2][0] = (*state)[2][2];
    (*state)[2][2] = temp;
    temp = (*state)[2][1];
    (*state)[2][1] = (*state)[2][3];
    (*state)[2][3] = temp;

    temp = (*state)[3][0];
    (*state)[3][0] = (*state)[3][1];
    (*state)[3][1] = (*state)[3][2];
    (*state)[3][2] = (*state)[3][3];
    (*state)[3][3] = temp;
}

__device__
void InvMixColumns(state_t* state) {
    int i, j;
    uint8_t col[4];
    for (j = 0; j < 4; j++) {
        for (i = 0; i < 4; i++) {
            col[i] = (*state)[i][j];
        }

        (*state)[0][j] = mult_x14[col[0]] ^ mult_x11[col[1]] ^ mult_x13[col[2]] ^ mult_x9[col[3]];
        (*state)[1][j] = mult_x9[col[0]] ^ mult_x14[col[1]] ^ mult_x11[col[2]] ^ mult_x13[col[3]];
        (*state)[2][j] = mult_x13[col[0]] ^ mult_x9[col[1]] ^ mult_x14[col[2]] ^ mult_x11[col[3]];
        (*state)[3][j] = mult_x11[col[0]] ^ mult_x13[col[1]] ^ mult_x9[col[2]] ^ mult_x14[col[3]];
    }
}

// 128 bit chunk of data - 16 chars
__device__
void AES_Decrypt_Block(uint8_t* cipherText, uint8_t* plainText, uint32_t* roundKeys, unsigned int numround) {
    state_t state;
    int i, j, d;
    for (i = 0, d = 0; i < 4; i++)
        for (j = 0; j < 4; j++, d++) {
            state[j][i] = (uint8_t) cipherText[d];
        }
    AddRoundKey(&state, roundKeys, numround);
    InvShiftRows(&state);
    InvSubBytes(&state);

    // the rest of the rounds except the final
    int round;
    for (round = numround-1; round > 1; round--) {
        AddRoundKey(&state, roundKeys, round);
        InvMixColumns(&state);
        InvShiftRows(&state);
        InvSubBytes(&state);
    }

    AddRoundKey(&state, roundKeys, 1);

    for (i = 0, d = 0; i < 4; i++)
        for (j = 0; j < 4; j++, d++) {
            plainText[d] = (uint8_t) state[j][i];
        }
}

void getDecKeyfromAsciiKey(char* asciiKey, uint32_t* decimalKey, uint32_t keyLength_words)
{
   char word[NUM_CHARS_IN_WORD];

   for(int i = 0; i < keyLength_words; i++)
   {
        strncpy(word, asciiKey+(i*NUM_CHARS_IN_WORD), NUM_CHARS_IN_WORD);
        decimalKey[i] = (uint32_t)strtoul(word, NULL, BASE_HEX);
   }

   return;
}